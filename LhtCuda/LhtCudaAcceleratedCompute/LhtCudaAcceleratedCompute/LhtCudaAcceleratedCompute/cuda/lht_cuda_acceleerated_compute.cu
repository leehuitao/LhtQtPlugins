#include "lht_cuda_acceleerated_compute.cuh"
#include "hip/hip_runtime.h"
#include ""


void LhtComputeInit()
{

}

//向量排序  单精度浮点数Ascend  = true 升序   false  降序
bool LhtSortFloat(float* in1, int len, float* result, bool Ascend , bool needToCopyToTheDevice)
{
	float* temp;
	hipMalloc(&result, sizeof(float) * len);
	if (needToCopyToTheDevice) {
		hipMalloc(&temp, sizeof(float) * len);
		hipMemcpy(in1, temp, sizeof(float)*len, hipMemcpyHostToDevice);
	}
	else {
		temp = in1;
	}


	return 1;
}

//向量点乘运算  单精度浮点数
bool LhtDotProductFloat(float* in1, float* in2, int len, float* result, bool needToCopyToTheDevice)
{
	float* temp1;
	float* temp2;
	hipMalloc(&result, sizeof(float) * len);
	if (needToCopyToTheDevice) {
		hipMalloc(&temp1, sizeof(float) * len);
		hipMemcpy(in1, temp1, sizeof(float) * len, hipMemcpyHostToDevice);
		hipMalloc(&temp2, sizeof(float) * len);
		hipMemcpy(in1, temp2, sizeof(float) * len, hipMemcpyHostToDevice);
	}
	else {
		temp1 = in1;
		temp2 = in2;
	}


	return 1;
}

//向量对单个值逐元素乘法  单精度浮点数
bool LhtMulCFloat(float* in1, float in2, int len, float* result, bool needToCopyToTheDevice)
{
	float* temp;
	hipMalloc(&result, sizeof(float) * len);
	if (needToCopyToTheDevice) {
		hipMalloc(&temp, sizeof(float) * len);
		hipMemcpy(in1, temp, sizeof(float) * len, hipMemcpyHostToDevice);
	}
	else {
		temp = in1;
	}


	return 1;
}
//复数乘法
bool LhtMulCFloat(LhtComplex* in1, LhtComplex* in2, int len, LhtComplex* result, bool needToCopyToTheDevice)
{

}
//向量加法  单精度浮点数
bool LhtAddFloat(float* in1, float* in2, int len, float* result, bool needToCopyToTheDevice)
{
	float* temp;
	hipMalloc(&result, sizeof(float) * len);
	if (needToCopyToTheDevice) {
		hipMalloc(&temp, sizeof(float) * len);
		hipMemcpy(in1, temp, sizeof(float) * len, hipMemcpyHostToDevice);
	}
	else {
		temp = in1;
	}


	return 1;
}
//向量减法  单精度浮点数
bool LhtSubFloat(float* in1, float* in2, int len, float* result, bool needToCopyToTheDevice)
{
	float* temp;
	hipMalloc(&result, sizeof(float) * len);
	if (needToCopyToTheDevice) {
		hipMalloc(&temp, sizeof(float) * len);
		hipMemcpy(in1, temp, sizeof(float) * len, hipMemcpyHostToDevice);
	}
	else {
		temp = in1;
	}


	return 1;
}
//向量和  
bool LhtSumFloat(float* in1, int len, float* result, bool needToCopyToTheDevice)
{
	float* temp;
	hipMalloc(&result, sizeof(float) * len);
	if (needToCopyToTheDevice) {
		hipMalloc(&temp, sizeof(float) * len);
		hipMemcpy(in1, temp, sizeof(float) * len, hipMemcpyHostToDevice);
	}
	else {
		temp = in1;
	}


	return 1;
}