#include "hip/hip_runtime.h"
#include "lht_basic_kernel.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

//点加
__global__ void lhtAdd(float* in1, float* in2, float* out, int len)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < len)
		out[index] = in1[index] + in2[index];
}
//点减
__global__ void lhtSub(float* in1, float* in2, float* out, int len)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < len)
		out[index] = in1[index] - in2[index];
}
//点乘
__global__ void lhtMulc(float* in1, float* in2, float* out, int len)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < len)
		out[index] = in1[index] * in2[index];
}

//乘系数
__global__ void lhtMulc(float* in1, float int2, float* out, int len)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < len)
		out[index] = in1[index] * int2;
}

//复数乘
__global__ void lhtMulc(hipfftComplex* in1, hipfftComplex* in2, hipfftComplex* out, int len)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < len) {
		//float a = in1[index].x; // 实部
		//float b = in1[index].y; // 虚部
		//float c = in2[index].x; // 实部
		//float d = in2[index].y; // 虚部
		out[index].x = in1[index].x * in2[index].x - in1[index].y * in2[index].y; // 结果的实部
		out[index].y = in1[index].x * in2[index].y + in1[index].y * in2[index].x; // 结果的虚部
	}
}

//排序
__global__ void lhtSort(float* in1, float* out, int len)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
}

//向量和  包含下面的均值都是规约求和
__global__ void lhtSum(float* in1, float* out, int len)
{
	unsigned int tid = threadIdx.x;
	//boundary check
	if (tid >= len) return;
	//convert global data pointer to the 
	float* idata = in1 + blockIdx.x * blockDim.x;
	//in-place reduction in global memory
	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		if ((tid % (2 * stride)) == 0)
		{
			idata[tid] += idata[tid + stride];
		}
		//synchronize within block
		__syncthreads();
	}
	//write result for this block to global mem
	if (tid == 0)
		out[blockIdx.x] = idata[0];
}

//向量绝对值的和
__global__ void lhtAbsSum(float* in1, float* out, int len)
{
	unsigned int tid = threadIdx.x;
	//boundary check
	if (tid >= len) return;
	//convert global data pointer to the 
	float* idata = in1 + blockIdx.x * blockDim.x;
	//in-place reduction in global memory
	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		if ((tid % (2 * stride)) == 0)
		{
			idata[tid] += (idata[tid + stride] > 0 ? idata[tid + stride] : -idata[tid + stride]);
		}
		//synchronize within block
		__syncthreads();
	}
	//write result for this block to global mem
	if (tid == 0)
		out[blockIdx.x] = idata[0];
}
//向量均值
__global__ void lhtMean(float* in1, float* out, int len)
{
	int blocksize = 1024;
	float gpu_sum = 0;
	dim3 block(blocksize, 1);
	dim3 grid((len - 1) / block.x + 1, 1);
	lhtSum(in1, out, len);
	for (int i = 0; i < grid.x; i++)
		gpu_sum += out[i];
	out[0] = gpu_sum / len;
}

//向量绝对值均值
__global__ void lhtAbsMean(float* in1, float* out, int len)
{
	int blocksize = 1024;
	float gpu_sum = 0;
	dim3 block(blocksize, 1);
	dim3 grid((len - 1) / block.x + 1, 1);
	lhtSum(in1, out, len);
	for (int i = 0; i < grid.x; i++)
		gpu_sum += out[i];
	out[0] = gpu_sum / len;
}

//向量最大值
__global__ void lhtMax(float* in1, float* out, int len)
{
	int tid = threadIdx.x;
	if (tid >= len)return;

	float* data = in1 + blockIdx.x * blockDim.x;
	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		if ((tid % (2 * stride)) == 0)
		{
			if (data[tid] < data[tid + stride])
			{
				data[tid] = data[tid + stride];
			}
		}
		__syncthreads();
	}
	if (tid == 0)
	{
		out[blockIdx.x] = data[0];
	}
}

//向量最小值
__global__ void lhtMin(float* in1, float* out, int len)
{
	int tid = threadIdx.x;
	if (tid >= len)return;

	float* data = in1 + blockIdx.x * blockDim.x;
	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		if ((tid % (2 * stride)) == 0)
		{
			if (data[tid] < data[tid + stride])
			{
				data[tid] = data[tid + stride];
			}
		}
		__syncthreads();
	}
	if (tid == 0)
	{
		out[blockIdx.x] = data[0];
	}
}



void LhtCopy2Host(float* in1, float* out, int len)
{
	hipMemcpy(out, in1, sizeof(float)*len, hipMemcpyDeviceToHost);
}