﻿
#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>


//结果检查
void checkResult(float* hostRef, float* gpuRef, double epsilon, const int N)  // 检查结果 epsilon 允许误差范围
{
    bool match = 1;  // 定义匹配标志
    for (int i = 0; i < N; i++)  // 比较每个元素
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)  // 如果误差超过范围
        {
            match = 0;  // 匹配标志置0
            printf("Arrays do not match!\n");  // 打印提示信息
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);  // 打印不匹配的元素
            break;
        }
    }
    if (match) printf("Arrays match.\n\n");  // 如果匹配，打印提示信息
}


int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus; //= addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

